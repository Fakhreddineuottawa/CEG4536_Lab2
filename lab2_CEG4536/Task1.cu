#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Addition des vecteurs en parallèle.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda a échoué !");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset doit être appelé avant de quitter afin que les outils de profilage
    // et de traçage comme Nsight et Visual Profiler montrent des traces complètes.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset a échoué !");
        return 1;
    }

    return 0;
}

// Fonction d'assistance pour utiliser CUDA pour additionner les vecteurs en parallèle.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choisir quel GPU utiliser, modifier ceci sur un système multi-GPU.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice a échoué ! Avez-vous un GPU compatible CUDA ?");
        goto Error;
    }

    // Allouer des tampons GPU pour les trois vecteurs (deux d'entrée, un de sortie).
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc a échoué !");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc a échoué !");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc a échoué !");
        goto Error;
    }

    // Copier les vecteurs d'entrée de la mémoire hôte aux tampons GPU.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy a échoué !");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy a échoué !");
        goto Error;
    }

    // Lancer un noyau sur le GPU avec un thread par élément.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Vérifier les erreurs lors du lancement du noyau
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Le lancement de addKernel a échoué : %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize attend que le noyau se termine, et retourne
    // toute erreur rencontrée durant le lancement.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize a retourné le code d'erreur %d après le lancement de addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copier le vecteur de sortie du tampon GPU vers la mémoire hôte.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy a échoué !");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
