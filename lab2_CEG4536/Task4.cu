#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <iostream>

__global__ void reductionKernel(int* input, int* output, int size) {
    extern __shared__ int sharedData[];
    int tid = threadIdx.x;
    int globalIndex = blockIdx.x * blockDim.x * 2 + tid;

    // Charger les données dans la mémoire partagée avec vérification stricte des limites
    sharedData[tid] = (globalIndex < size) ? input[globalIndex] : 0;
    if (globalIndex + blockDim.x < size) {
        sharedData[tid] += input[globalIndex + blockDim.x];
    }
    __syncthreads();

    // Effectuer la réduction dans la mémoire partagée
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    // Écrire le résultat de ce bloc dans le tableau de sortie
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

int main() {
    const int size = 1024; // Nombre total d'éléments
    const int bytes = size * sizeof(int);
    const int blockSize = 256; // Nombre de threads par bloc
    const int gridSize = (size + blockSize * 2 - 1) / (blockSize * 2); // Nombre de blocs

    int* h_input = new int[size];
    int* h_output = new int[gridSize];
    int* d_input, * d_output;

    // Initialiser le tableau d'entrée avec des 1
    for (int i = 0; i < size; i++) {
        h_input[i] = 1;
    }

    // Allouer la mémoire sur le périphérique
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, gridSize * sizeof(int));
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // Lancer le noyau de réduction
    reductionKernel << <gridSize, blockSize, blockSize * sizeof(int) >> > (d_input, d_output, size);
    hipDeviceSynchronize();

    // Copier le résultat de retour vers l'hôte
    hipMemcpy(h_output, d_output, gridSize * sizeof(int), hipMemcpyDeviceToHost);

    // Effectuer la réduction finale sur le CPU
    int finalSum = 0;
    for (int i = 0; i < gridSize; i++) {
        finalSum += h_output[i];
    }

    // S'il reste des éléments qui n'ont pas été traités dans la réduction GPU, les additionner sur le CPU
    int remainingElementsSum = 0;
    for (int i = gridSize * blockSize * 2; i < size; i++) {
        remainingElementsSum += h_input[i];
    }
    finalSum += remainingElementsSum;

    // Afficher le résultat final
    std::cout << "Somme finale après réduction : " << finalSum << std::endl;

    // Libérer la mémoire
    delete[] h_input;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
